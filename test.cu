
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);

    printf("GPU Architecture: sm_%d%d\n", prop.major, prop.minor);

    return 0;
}
